#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaScan.cuh"
#include <Psapi.h>
#pragma comment(lib,"Psapi.lib")
#include <stdio.h>

#define dwStartAddr 0x00000000
#define dwEndAddr 0x7fffffff
HANDLE hProcess = INVALID_HANDLE_VALUE;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


#define SAFE_RELEASE(p) { if(p)!=hipSuccess { (p)->Release(); (p)=NULL; } } 

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__global__ int CudaScan(BYTE* pMemoryBuffer, int dwBufferSize, BYTE* bstr, DWORD dwStrLen)
{
	BYTE* pCurrMemoryData = NULL;
	MEMORY_BASIC_INFORMATION	mbi;
	std::vector<MEMORY_REGION> m_vMemoryRegion;
	mbi.RegionSize = 0x1000;
	DWORD dwAddress = dwStartAddr;
	int* HostBuff = 0;
	int* DeviceBuff = 0;
	hipError_t CopyStatus;

	//�����ڴ��,�ŵ�vector������
	
	if (!hProcess) return -1;
	while (VirtualQueryEx(hProcess, (LPCVOID)dwAddress, &mbi, sizeof(mbi)) && (dwAddress < dwEndAddr) && ((dwAddress + mbi.RegionSize) > dwAddress))
	{
		if ((mbi.State == MEM_COMMIT) && ((mbi.Protect & PAGE_GUARD) == 0) && (mbi.Protect != PAGE_NOACCESS) && ((mbi.AllocationProtect & PAGE_NOCACHE) != PAGE_NOCACHE))
		{
			CopyStatus = hipMalloc((void**)& DeviceBuff, sizeof(mbi));
			SAFE_RELEASE(CopyStatus);
			MEMORY_REGION mData = { 0 };
			mData.dwBaseAddr = (DWORD_PTR)mbi.BaseAddress;
			mData.dwMemorySize = mbi.RegionSize;
			m_vMemoryRegion.push_back(mData);
			cudaStatus = hipMemcpy(DeviceBuff, mbi, size * sizeof(int), hipMemcpyHostToDevice);
			
		}
		dwAddress = (DWORD)mbi.BaseAddress + mbi.RegionSize;
	}

	//�����ڴ��
	std::vector<MEMORY_REGION>::iterator it;
	for (it = m_vMemoryRegion.begin(); it != m_vMemoryRegion.end(); it++)
	{
		MEMORY_REGION mData = *it;

		//����ǵ�ǰ���̣�ֱ����ָ��
		DWORD_PTR dwNumberOfBytesRead = 0;
		if (bIsCurrProcess)
		{
			pCurrMemoryData = (BYTE*)mData.dwBaseAddr;
			dwNumberOfBytesRead = mData.dwMemorySize;
		}
		else
		{
			pCurrMemoryData = new BYTE[mData.dwMemorySize];//��ö���������ģ���������ڴ治�ᱻ����
			ZeroMemory(pCurrMemoryData, mData.dwMemorySize);
			ReadProcessMemory(hProcess, (LPCVOID)mData.dwBaseAddr, pCurrMemoryData, mData.dwMemorySize, &dwNumberOfBytesRead);
			if ((int)dwNumberOfBytesRead <= 0)
			{
				delete[] pCurrMemoryData;
				continue;
			}
		}
		if (iSearchMode == 0)
		{
			DWORD_PTR dwOffset = 0;
			int iOffset = MemFind(pCurrMemoryData, dwNumberOfBytesRead, bSearchData, nSearchSize);
			while (iOffset != -1)
			{
				dwOffset += iOffset;
				vRet.push_back(dwOffset + mData.dwBaseAddr);
				dwOffset += nSearchSize;
				iOffset = MemFind(pCurrMemoryData + dwOffset, dwNumberOfBytesRead - dwOffset - nSearchSize, bSearchData, nSearchSize);
			}
		}
		else if (iSearchMode == 1)
		{

			DWORD_PTR dwOffset = 0;
			int iOffset = SundaySearch(pCurrMemoryData, dwNumberOfBytesRead, bSearchData, nSearchSize);
			while (iOffset != -1)
			{
				dwOffset += iOffset;
				vRet.push_back(dwOffset + mData.dwBaseAddr);
				dwOffset += nSearchSize;
				iOffset = MemFind(pCurrMemoryData + dwOffset, dwNumberOfBytesRead - dwOffset - nSearchSize, bSearchData, nSearchSize);
			}

		}
		//������ǵ�ǰ���̣���Ҫ�ͷ�
		if (!bIsCurrProcess && (pCurrMemoryData != NULL))
		{
			delete[] pCurrMemoryData;
			pCurrMemoryData = NULL;
		}

	}
	return TRUE;
}


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
